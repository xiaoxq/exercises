
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5

__global__
void gpu_local_memory(int d_in) {
  int t_local = d_in * threadIdx.x;
  printf("Value of Local variable in current thread is: %d\n", t_local);
}

int main(int argc, char **argv) {
  gpu_local_memory <<<1, N>>>(5);
  hipDeviceSynchronize();
  return 0;
}

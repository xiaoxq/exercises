#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define BLOCKS 10
#define THREADS 10

// Define texture reference for 1-d access
texture <float, 1, hipReadModeElementType> textureRef;

__global__
void gpu_texture_memory(int n, float *d_out) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    d_out[idx] = tex1D(textureRef, float(idx));
  }
}

int main() {
  const size_t N = BLOCKS * THREADS;
  const size_t kMemSize = sizeof(float) * N;
  float *d_out;
  hipMalloc((void**)&d_out, kMemSize);

  // Allocate space on the host for the results
  float h_in[N];
  for (int i = 0; i < N; i++) {
    h_in[i] = float(i);
  }
  float *h_out = (float*)malloc(kMemSize);

  // Define CUDA Array
  hipArray *cu_Array;
  hipMallocArray(&cu_Array, &textureRef.channelDesc, N, 1);
  hipMemcpyToArray(cu_Array, 0, 0, h_in, kMemSize, hipMemcpyHostToDevice);

  // Bind a texture to the CUDA array
  hipBindTextureToArray(textureRef, cu_Array);
  gpu_texture_memory <<<BLOCKS, THREADS>>>(N, d_out);

  hipMemcpy(h_out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    std::cout << "Average between two nearest elements is: "
              << h_out[i] << std::endl;
  }
  free(h_out);
  hipFree(d_out);
  hipFreeArray(cu_Array);
  hipUnbindTexture(textureRef);
  return 0;
}

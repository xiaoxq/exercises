
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKS 100
#define THREADS 100
#define ARRAY_SIZE 10

__global__
void gpu_increment_without_atomic(int *d_a) {
  const int tid = (blockIdx.x * blockDim.x + threadIdx.x) % ARRAY_SIZE;
  d_a[tid] += 1;
}

__global__
void gpu_increment_atomic(int *d_a) {
  const int tid = (blockIdx.x * blockDim.x + threadIdx.x) % ARRAY_SIZE;
  atomicAdd(&d_a[tid], 1);
}

int main(int argc, char **argv) {
  const int MEM_SIZE = ARRAY_SIZE * sizeof(int);

  int h_a[ARRAY_SIZE];
  int* d_a;
  hipMalloc((void **)&d_a, MEM_SIZE);

  // Calculate without atomic.
  hipMemset((void *)d_a, 0, MEM_SIZE);
  gpu_increment_without_atomic <<<BLOCKS, THREADS>>>(d_a);
  hipMemcpy(h_a, d_a, MEM_SIZE, hipMemcpyDeviceToHost);
  std::cout << "Without atomic:" << std::endl;
  for (int i = 0; i < ARRAY_SIZE; i++) {
    std::cout << "h_a[" << i << "] = " << h_a[i] << std::endl;
  }

  // Calculate with atomic.
  hipMemset((void *)d_a, 0, MEM_SIZE);
  gpu_increment_atomic <<<BLOCKS, THREADS>>>(d_a);
  hipMemcpy(h_a, d_a, MEM_SIZE, hipMemcpyDeviceToHost);
  std::cout << "With atomic:" << std::endl;
  for (int i = 0; i < ARRAY_SIZE; i++) {
    std::cout << "h_a[" << i << "] = " << h_a[i] << std::endl;
  }

  hipFree(d_a);
  return 0;
}
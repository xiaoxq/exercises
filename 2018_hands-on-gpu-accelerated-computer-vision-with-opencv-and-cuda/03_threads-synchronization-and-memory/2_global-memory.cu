
#include <hip/hip_runtime.h>
#include <iostream>

#define N 5

__global__
void gpu_global_memory(int *d_a) {
  d_a[threadIdx.x] = threadIdx.x;
}

int main(int argc, char **argv) {
  int h_a[N];
  int *d_a;

  const size_t kMemSize = sizeof(int) * N;
  hipMalloc((void **)&d_a, kMemSize);
  hipMemcpy((void *)d_a, (void *)h_a, kMemSize, hipMemcpyHostToDevice);

  gpu_global_memory <<<1, N>>>(d_a);
  hipMemcpy((void *)h_a, (void *)d_a, kMemSize, hipMemcpyDeviceToHost);

  std::cout  << "Array in Global Memory is:" << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout  << "At Index: " << i << " --> " << h_a[i] << std::endl;
  }
  return 0;
}

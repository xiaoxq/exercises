#include <hip/hip_runtime.h>


#include <iostream>

// Defining two constants
__constant__ int constant_f;
__constant__ int constant_g;

#define N 5

__global__
void gpu_constant_memory(float *d_in, float *d_out) {
  // Getting thread index for current kernel
  const int tid = threadIdx.x; 
  d_out[tid] = constant_f * d_in[tid] + constant_g;
}

int main(void) {
  // Copy constants to constant memory
  int h_f = 2, h_g = 20;
  hipMemcpyToSymbol(HIP_SYMBOL(constant_f), &h_f, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(constant_g), &h_g, sizeof(int));
  
  float *d_in, *d_out;
  // Allocate the memory on the cpu.
  const size_t kMemSize = N * sizeof(float);
  hipMalloc((void**)&d_in, kMemSize);
  hipMalloc((void**)&d_out, kMemSize);

  // Copy Array from host to device.
  const float h_in[N] = {0, 1, 2, 3, 4};
  hipMemcpy(d_in, h_in, kMemSize, hipMemcpyHostToDevice);
  gpu_constant_memory <<<1, N>>>(d_in, d_out);

  // Coping result back to host from device memory
  float h_out[N];
  hipMemcpy(h_out, d_out, kMemSize, hipMemcpyDeviceToHost);
  
  // Printing result on console
  for (int i = 0; i < N; i++) {
    std::cout << "h_in / h_out[" << i << "] = " << h_in[i] << " / " << h_out[i]
              << std::endl;
  }

  hipFree(d_in);
  hipFree(d_out);
  return 0;
}
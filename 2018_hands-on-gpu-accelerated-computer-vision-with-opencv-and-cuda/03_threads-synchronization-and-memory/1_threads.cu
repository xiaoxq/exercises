#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>

// TODO(xiaoxq): Fix the crash.

// Number of elements in Array
#define N 1000000

// Definition of kernel function to add two variables
__global__
void gpuAdd(int *d_a, int *d_b, int *d_c) {
  // gridDim.x    Grid width (Number of blocks)
  // blockDim.x   Block width (Size of a block)
  // blockIdx.x   Index of current block
  // threadIdx.x  Index of current thread in current block
  printf("gridDim.x = %d, blockDim.x = %d, blockIdx.x = %d, threadIdx.x = %d\n",
         gridDim.x, blockDim.x, blockIdx.x, threadIdx.x);
  for (int grid = 0; ; ++grid) {
    const int tid = threadIdx.x + blockDim.x * (blockIdx.x + grid * gridDim.x);
    if (tid < N) {
      d_c[tid] = d_a[tid] + d_b[tid];
    } else {
      return;
    }
  }
}

int main(void) {
  // Declare host and device Arrays
  int h_a[N], h_b[N], h_c[N];
  for (int i = 0; i < N; i++) {
    h_a[i] = 2 * i * i;
    h_b[i] = i;
  }

  int *d_a, *d_b, *d_c;
  hipMalloc((void**)&d_a, N * sizeof(int));
  hipMalloc((void**)&d_b, N * sizeof(int));
  hipMalloc((void**)&d_c, N * sizeof(int));

  hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

  std::cout << "gpuAdd started." << std::endl;
  gpuAdd <<<512, 256>>>(d_a, d_b, d_c);
  hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  std::cout << "gpuAdd ended." << std::endl;

  bool correct = true;
  for (int i = 0; i < N; i++) {
    if ((h_a[i] + h_b[i] != h_c[i])) {
      correct = false;
      break;
    }
  }

  std::cout << (correct ? "Finished successfully!"
                        : "Finished with error!") << std::endl;

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}

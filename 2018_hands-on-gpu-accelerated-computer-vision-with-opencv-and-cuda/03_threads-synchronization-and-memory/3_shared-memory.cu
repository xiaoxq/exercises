
#include <hip/hip_runtime.h>
#include <iostream>

#define N 5

__global__
void gpu_shared_memory(float *d_a) {
  int i, index = threadIdx.x;
  float sum = 0.0f;

  // Defining shared memory.
  __shared__ float sh_arr[10];
  sh_arr[index] = d_a[index];

  // This directive ensure all the writes to shared memory have completed.
  __syncthreads();
  for (i = 0; i<= index; i++) { 
    sum += sh_arr[i]; 
  }
  const float average = sum / (index + 1.0f);
  d_a[index] = average;
  // This statement is redundant and will have no effect on overall execution.
  sh_arr[index] = 0;
}

int main(int argc, char **argv) {
  float h_a[10] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  float *d_a;

  // Allocate global memory on the device.
  const size_t kMemSize = sizeof(float) * 10;
  hipMalloc((void **)&d_a, kMemSize);
  hipMemcpy((void *)d_a, (void *)h_a, kMemSize, hipMemcpyHostToDevice);

  gpu_shared_memory <<<1, 10>>>(d_a);
  hipMemcpy((void *)h_a, (void *)d_a, kMemSize, hipMemcpyDeviceToHost);

  for (int i = 0; i < 10; i++) {
    std::cout << "The running average after " << i << " element is "
              << h_a[i] << std::endl;
  }
  return 0;
}

#include <hip/hip_runtime.h>


#include <iostream>

int main() {
  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, 0);

  int *d_a;
  const int MEM_SIZE = 2048;
  hipMalloc((void**)&d_a, MEM_SIZE);
  hipMemset((void *)d_a, 0, MEM_SIZE);

  hipDeviceSynchronize();
  hipEventRecord(e_stop, 0);
  hipEventSynchronize(e_stop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, e_start, e_stop);
  std::cout << "Elapsed time: " << elapsedTime << std::endl;
  return 0;
}

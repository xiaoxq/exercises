#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 1000
#define NUM_BIN 16

__global__
void histogram_atomic(int *d_b, int *d_a) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int item = d_a[tid];
  if (tid < SIZE) {
    atomicAdd(&(d_b[item]), 1);
  }
}

__global__
void histogram_shared_memory(int *d_b, int *d_a) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int offset = blockDim.x * gridDim.x;
  __shared__ int cache[256];
  cache[threadIdx.x] = 0;
  __syncthreads();

  while (tid < SIZE) {
    atomicAdd(&(cache[d_a[tid]]), 1);
    tid += offset;
  }
  __syncthreads();
  atomicAdd(&(d_b[threadIdx.x]), cache[threadIdx.x]);
}

int main() {
  int h_a[SIZE];
  for (int i = 0; i < SIZE; i++) {
    h_a[i] = i % NUM_BIN;
  }
  int h_b[NUM_BIN];
  for (int i = 0; i < NUM_BIN; i++) {
    h_b[i] = 0;
  }

  // declare GPU memory pointers
  int * d_a;
  int * d_b;

  // allocate GPU memory
  hipMalloc((void **)&d_a, SIZE * sizeof(int));
  hipMalloc((void **)&d_b, NUM_BIN * sizeof(int));

  // transfer the arrays to the GPU
  hipMemcpy(d_a, h_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NUM_BIN * sizeof(int), hipMemcpyHostToDevice);

  // launch the kernel
  // histogram_atomic << <((SIZE+NUM_BIN-1) / NUM_BIN), NUM_BIN >> >(d_b, d_a);
  histogram_shared_memory
  <<<((SIZE + NUM_BIN - 1) / NUM_BIN), NUM_BIN>>>(d_b, d_a);

  // copy back the sum from GPU
  hipMemcpy(h_b, d_b, NUM_BIN * sizeof(int), hipMemcpyDeviceToHost);
  printf("Histogram using 16 bin without shared Memory is: \n");
  for (int i = 0; i < NUM_BIN; i++) {
    printf("bin %d: count %d\n", i, h_b[i]);
  }

  // free GPU memory allocation
  hipFree(d_a);
  hipFree(d_b);
  return 0;
}

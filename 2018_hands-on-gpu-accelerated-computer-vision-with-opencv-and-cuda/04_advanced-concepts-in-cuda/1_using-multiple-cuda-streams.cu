#include <hip/hip_runtime.h>


#include <iostream>

// Defining number of elements in Array
#define N 50000

// Defining Kernel function for vector addition
__global__
void gpuAdd(int *d_a, int *d_b, int *d_c) {
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x;
       tid < N; tid += blockDim.x * gridDim.x) {
    d_c[tid] = d_a[tid] + d_b[tid];
  }
}

int main(void) {
  const size_t kSizeOfNInt = N * sizeof(int);

  // Defining host arrays
  int *h_a, *h_b, *h_c;
  hipHostAlloc((void**)&h_a, 2 * kSizeOfNInt, hipHostMallocDefault);
  hipHostAlloc((void**)&h_b, 2 * kSizeOfNInt, hipHostMallocDefault);
  hipHostAlloc((void**)&h_c, 2 * kSizeOfNInt, hipHostMallocDefault);
  for (int i = 0; i < N * 2; i++) {
    h_a[i] = 2 * i * i;
    h_b[i] = i;
  }

  // Defining device pointers for stream 0
  int *d_a0, *d_b0, *d_c0;
  hipMalloc((void**)&d_a0, kSizeOfNInt);
  hipMalloc((void**)&d_b0, kSizeOfNInt);
  hipMalloc((void**)&d_c0, kSizeOfNInt);

  // Defining device pointers for stream 1
  int *d_a1, *d_b1, *d_c1;
  hipMalloc((void**)&d_a1, kSizeOfNInt);
  hipMalloc((void**)&d_b1, kSizeOfNInt);
  hipMalloc((void**)&d_c1, kSizeOfNInt);

  // Init streams.
  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);

  // Asynchrnous Memory Copy Operation for both streams
  hipMemcpyAsync(d_a0, h_a , kSizeOfNInt, hipMemcpyHostToDevice, stream0);
  hipMemcpyAsync(d_a1, h_a + N, kSizeOfNInt, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(d_b0, h_b , kSizeOfNInt, hipMemcpyHostToDevice, stream0);
  hipMemcpyAsync(d_b1, h_b + N, kSizeOfNInt, hipMemcpyHostToDevice, stream1);

  // Kernel call
  gpuAdd <<<512, 512, 0, stream0>>> (d_a0, d_b0, d_c0);
  gpuAdd <<<512, 512, 0, stream1>>> (d_a1, d_b1, d_c1);

  // Copy result back to host memory from device memory
  hipMemcpyAsync(h_c , d_c0, kSizeOfNInt, hipMemcpyDeviceToHost, stream0);
  hipMemcpyAsync(h_c + N, d_c1, kSizeOfNInt, hipMemcpyDeviceToHost, stream0);

  hipDeviceSynchronize();
  hipStreamSynchronize(stream0);
  hipStreamSynchronize(stream1);

  // Check result.
  const int idx = N / 2;
  std::cout << "h_a[" << idx << "] = " << h_a[idx] << std::endl;
  std::cout << "h_b[" << idx << "] = " << h_b[idx] << std::endl;
  std::cout << "h_c[" << idx << "] = " << h_c[idx] << std::endl;

  // Free up memory
  hipFree(d_a0);
  hipFree(d_b0);
  hipFree(d_c0);
  hipFree(d_a0);
  hipFree(d_b0);
  hipFree(d_c0);
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);
  return 0;
}

#include <hip/hip_runtime.h>

#include <iostream>

#define SIZE 1000
#define NUM_BIN 16

__global__
void histogram_atomic(int *d_b, int *d_a) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int item = d_a[tid];
  if (tid < SIZE) {
    atomicAdd(&(d_b[item]), 1);
  }
}

__global__
void histogram_shared_memory(int *d_b, int *d_a) {
  __shared__ int cache[256];
  cache[threadIdx.x] = 0;
  __syncthreads();

  for (int tid = threadIdx.x + blockDim.x * blockIdx.x; tid < SIZE;
       tid += blockDim.x * gridDim.x) {
    atomicAdd(&(cache[d_a[tid]]), 1);
  }
  __syncthreads();
  atomicAdd(&(d_b[threadIdx.x]), cache[threadIdx.x]);
}

int main() {
  int h_a[SIZE];
  for (int i = 0; i < SIZE; i++) {
    h_a[i] = i % NUM_BIN;
  }
  int h_b[NUM_BIN];
  for (int i = 0; i < NUM_BIN; i++) {
    h_b[i] = 0;
  }

  // declare GPU memory pointers
  int *d_a;
  int *d_b;

  // allocate GPU memory
  hipMalloc((void **)&d_a, SIZE * sizeof(int));
  hipMalloc((void **)&d_b, NUM_BIN * sizeof(int));

  // transfer the arrays to the GPU
  hipMemcpy(d_a, h_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NUM_BIN * sizeof(int), hipMemcpyHostToDevice);

  // launch the kernel
  // histogram_atomic << <((SIZE+NUM_BIN-1) / NUM_BIN), NUM_BIN >> >(d_b, d_a);
  const int blocks = (SIZE + NUM_BIN - 1) / NUM_BIN;
  histogram_shared_memory <<<blocks, NUM_BIN>>> (d_b, d_a);

  // copy back the sum from GPU
  hipMemcpy(h_b, d_b, NUM_BIN * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < NUM_BIN; i++) {
    std::cout << "h_b[" << i << "] = " << h_b[i] << std::endl;
  }

  // free GPU memory allocation
  hipFree(d_a);
  hipFree(d_b);
  return 0;
}

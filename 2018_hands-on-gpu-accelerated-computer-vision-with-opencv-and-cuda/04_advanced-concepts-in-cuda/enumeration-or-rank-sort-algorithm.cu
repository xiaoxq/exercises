#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define arraySize 5
#define threadPerBlock 5

// Kernel Function for Rank sort
__global__
void addKernel(int *d_in, int *d_out) {
  int count = 0;
  int tid = threadIdx.x;
  int ttid = blockIdx.x * threadPerBlock + tid;
  int val = d_in[ttid];
  __shared__ int cache[threadPerBlock];
  for (int i = tid; i < arraySize; i += threadPerBlock) {
    cache[tid] = d_in[i];
    __syncthreads();
    for (int j = 0; j < threadPerBlock; ++j) {
      if (val > cache[j]) {
        ++count;
      }
    }
    __syncthreads();
  }
  d_out[count] = val;
}

int main() {
  const size_t kMemSize = arraySize * sizeof(int);
  // Define Host and Device Array
  int h_in[arraySize] = {5, 9, 3, 4, 8};
  int h_out[arraySize];

  // Allocate Memory on the device 
  int *d_in, *d_out;
  hipMalloc((void**)&d_in, kMemSize);
  hipMalloc((void**)&d_out, kMemSize);

  // Copy input vector from host memory to device memory.
  hipMemcpy(d_in, h_in, kMemSize, hipMemcpyHostToDevice);

  // Launch a kernel on the GPU with one thread for each element.
  addKernel <<<arraySize / threadPerBlock, threadPerBlock>>>(d_in, d_out);

  // Wait for device to finish operations
  hipDeviceSynchronize();
  // Copy output vector from GPU buffer to host memory.
  hipMemcpy(h_out, d_out, kMemSize, hipMemcpyDeviceToHost);
  printf("The Enumeration sorted Array is: \n");
  for (int i = 0; i < arraySize; i++) {
    printf("%d\n", h_b[i]);
  }
  // Free up device memory
  hipFree(d_in);
  hipFree(d_out);
  return 0;
}

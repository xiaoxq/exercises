#include <hip/hip_runtime.h>

#include <iostream>

#define ARRAY_SIZE 5
#define THREADS_PER_BLOCK 5
#define BLOCKS (ARRAY_SIZE / THREADS_PER_BLOCK)


// Kernel Function for Rank sort.
__global__
void addKernel(int *d_in, int *d_out) {
  int count = 0;
  const int val = d_in[blockIdx.x * THREADS_PER_BLOCK + threadIdx.x];
  __shared__ int cache[THREADS_PER_BLOCK];
  for (int i = threadIdx.x; i < ARRAY_SIZE; i += THREADS_PER_BLOCK) {
    cache[threadIdx.x] = d_in[i];
    __syncthreads();
    for (int j = 0; j < THREADS_PER_BLOCK; ++j) {
      if (val > cache[j]) {
        ++count;
      }
    }
    __syncthreads();
  }
  d_out[count] = val;
}

int main() {
  const size_t kMemSize = ARRAY_SIZE * sizeof(int);
  // Define Host and Device Array.
  int h_in[ARRAY_SIZE] = {5, 9, 3, 4, 8};
  int h_out[ARRAY_SIZE];

  // Allocate Memory on the device.
  int *d_in, *d_out;
  hipMalloc((void**)&d_in, kMemSize);
  hipMalloc((void**)&d_out, kMemSize);

  // Copy input vector from host memory to device memory.
  hipMemcpy(d_in, h_in, kMemSize, hipMemcpyHostToDevice);

  // Launch a kernel on the GPU with one thread for each element.
  addKernel <<<BLOCKS, ARRAY_SIZE / BLOCKS>>>(d_in, d_out);

  // Wait for device to finish operations.
  hipDeviceSynchronize();

  // Copy output vector from GPU buffer to host memory.
  hipMemcpy(h_out, d_out, kMemSize, hipMemcpyDeviceToHost);
  for (size_t i = 0; i < ARRAY_SIZE; ++i) {
    std::cout << h_out[i] << std::endl;
  }

  // Free up device memory.
  hipFree(d_in);
  hipFree(d_out);
  return 0;
}

#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>

using namespace std;

void GeneralDeviceProperties(const int device) {
  cout << "===========================================================" << endl;
  hipDeviceProp_t device_Property;
  hipGetDeviceProperties(&device_Property, device);
  cout << "Device " << device << ": " << device_Property.name << endl;

  int driver_Version, runtime_Version;
  hipDriverGetVersion(&driver_Version);
  hipRuntimeGetVersion(&runtime_Version);
  cout << "CUDA Driver Version: "
       << driver_Version / 1000 << "." << (driver_Version % 100) / 10 << endl;
  cout << "CUDA Runtime Version: "
       << runtime_Version / 1000 << "." << (runtime_Version % 100) / 10 << endl;

  const int mem_mb = device_Property.totalGlobalMem >> 20;
  cout << "Total amount of global memory: " << mem_mb << "MB." << endl;

  cout << "Multiprocessors: " << device_Property.multiProcessorCount << endl;
  cout << "GPU Clock rate: " << device_Property.clockRate * 1e-6f << "GHz" << endl;
  cout << "===========================================================" << endl;
}

int main(void) {
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  cout << "CUDA Capable device(s): " << device_count << endl;

  for (int i = 0; i < device_count; ++i) {
    GeneralDeviceProperties(i);
  }
  return 0;
}

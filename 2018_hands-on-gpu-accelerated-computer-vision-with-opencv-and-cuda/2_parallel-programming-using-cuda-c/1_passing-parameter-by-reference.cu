#include <iostream>

#include <hip/hip_runtime.h>


// Kernel function to add two variables, parameters are passed by reference
__global__
void gpuAdd(int *d_a, int *d_b, int *d_c) {
  *d_c = *d_a + *d_b;
}

int main() {
  // Defining host and variables
  int h_a = 1, h_b = 2, h_c;
  int *d_a, *d_b, *d_c;
  // Allocating memory for Device Pointers
  hipMalloc((void**)&d_a, sizeof(int));
  hipMalloc((void**)&d_b, sizeof(int));
  hipMalloc((void**)&d_c, sizeof(int));
  // Coping value of host variables in device memory
  hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

  // Calling kernel with one thread and one block with parameters passed by reference
  gpuAdd<<<1, 1>>>(d_a, d_b, d_c);
  // Coping result from device memory to host
  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  std::cout << "Passing Parameter by Reference Output: "
            << h_a << " + " << h_b << " = " << h_c << std::endl;

  // Free up memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}

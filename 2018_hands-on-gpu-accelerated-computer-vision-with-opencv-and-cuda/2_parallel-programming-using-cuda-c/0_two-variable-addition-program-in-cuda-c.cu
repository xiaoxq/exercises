#include <hip/hip_runtime.h>

#include <iostream>

// Definition of kernel function to add two variables
__global__
void gpuAdd(int d_a, int d_b, int *d_c) {
  *d_c = d_a + d_b;
}

int main() {
  //Defining host variable to store answer
  int h_c;
  //Defining device pointer
  int *d_c;

  //Allocating memory for device pointer
  hipMalloc((void**)&d_c, sizeof(int));

  //Kernel call by passing 1 and 4 as inputs and storing answer in d_c
  //<< <1,1> >> means 1 block is executed with 1 thread per block
  gpuAdd <<<1, 1>>>(1, 4, d_c);

  //Copy result from device memory to host memory
  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  std::cout << "1 + 4 = " << h_c << std::endl;

  //Free up memory
  hipFree(d_c);
  return 0;
}

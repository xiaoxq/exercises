
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__
void myfirstkernel(void) {
  // blockIdx.x gives the block number of current kernel
  printf("Hello! I'm thread in block: %d\n", blockIdx.x);
}

int main() {
  // A kernel call with 16 blocks and 1 thread per block
  myfirstkernel <<<16, 1>>>();

  // Function used for waiting for all kernels to finish
  hipDeviceSynchronize();

  std::cout << "All threads are finished!" << std::endl;
  return 0;
}

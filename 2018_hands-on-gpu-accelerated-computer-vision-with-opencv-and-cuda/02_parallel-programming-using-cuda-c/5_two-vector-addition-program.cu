#include <hip/hip_runtime.h>

#include <iostream>

//Defining number of elements in Array
#define N 5

//Defining Kernel function for vector addition
__global__
void gpuAdd(int *d_a, int *d_b, int *d_c) {
  // Getting block index of current kernel
  int tid = blockIdx.x;  // handle the data at this index
  if (tid < N) {
    d_c[tid] = d_a[tid] + d_b[tid];
  }
}

int main() {
  int h_a[N], h_b[N], h_c[N];
  for (int i = 0; i < N; i++) {
    h_a[i] = 2 * i * i;
    h_b[i] = i ;
  }

  int *d_a, *d_b, *d_c;
  hipMalloc((void**)&d_a, N * sizeof(int));
  hipMalloc((void**)&d_b, N * sizeof(int));
  hipMalloc((void**)&d_c, N * sizeof(int));

  // Copy data, do the calculation, and copy back.
  hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
  gpuAdd <<<N, 1>>>(d_a, d_b, d_c);
  hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

  // Printing result on console
  for (int i = 0; i < N; i++) {
    std::cout << i << ": " << h_a[i] << " + " << h_b[i] << " = " << h_c[i]
              << std::endl;
  }

  // Free up memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
